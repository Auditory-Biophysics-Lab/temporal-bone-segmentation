#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cutil.h>
#include <memcpy.cu>
#include <cubicPrefilter3D.cu>
#include <cubicTex3D.cu>

#include "cudaresize.h"

texture<float, hipTextureType3D, hipReadModeElementType> texRef;

__global__ 
void transformKernel(float *output, float wr, float hr, float dr, 
                     unsigned int width, unsigned int height, unsigned int depth, 
                     unsigned int zoffset) {
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    float u = ((float)x)*wr + 0.5f;
    float v = ((float)y)*hr + 0.5f;
    float w = ((float)(z + zoffset))*dr + 0.5f;

    // Read from texture and write to global memory
    if (x < width && y < height && z < depth)
        output[z*width*height + y*width + x] = tex3D(texRef, u, v, w);
}

template<class T> 
void interpolate_template(
        const T *input_array, T *output_array, unsigned int width, unsigned int height, unsigned int depth,
        unsigned int new_width, unsigned int new_height, unsigned int new_depth
    ) {
    // Load the input volume into the device memory; this also converts it to a float
    hipPitchedPtr bsplineCoeffs = CastVolumeHostToDevice(input_array, width, height, depth);

    // Run the pre-filter
    CubicBSplinePrefilter3D((float*)bsplineCoeffs.ptr, bsplineCoeffs.pitch, width, height, depth);

    // Now, we need to copy everything into a CUDA array. Ideally, this should be done entirely on 
    // the device, but for large input volumes we may run out of GPU memory so we'll have to copy 
    // it back to host RAM and then to the GPU again

    // bsplineCoeffs.pitch is the actual size of each row, including GPU-specific padding bytes 
    // (for performance reasons). This is the total size of the memory allocated for the 
    // bsplineCoeffs array, and should be the same as required for the array
    size_t needed = bsplineCoeffs.pitch*sizeof(float)*height*depth;

    // Read the total available GPU memory
    size_t free_mem, total;
    hipMemGetInfo(&free_mem, &total);
    printf("free_mem: %zu, total: %zu, needed: %zu\n", free_mem, total, needed);

    bool enough_gpu = free_mem < (needed + (needed >> 3));

    // Add ~12% to be safe
    if (true || !enough_gpu) {
        // Allocate a temp array on the host
        float *temp = (float *)malloc(sizeof(float)*width*height*depth);
        // Create a pitched pointer for CUDA (no padding bytes)
        hipPitchedPtr temp_ptr = make_hipPitchedPtr((void*)temp, width*sizeof(float), width, height);

        // Create the memcpy parameters
        hipMemcpy3DParms p = {0};
        // Extent width is in bytes for linear memory
        p.extent = make_hipExtent(width*sizeof(float), height, depth);
        p.srcPtr = bsplineCoeffs;
        p.dstPtr = temp_ptr;
        p.kind = hipMemcpyDeviceToHost;

        // Run the transfer
        hipMemcpy3D(&p);
        
        hipFree(bsplineCoeffs.ptr);

        // Replace the pointer
        bsplineCoeffs = temp_ptr;
    }
        
    hipArray *cuArray = 0;
    // Extent width is in elements for array memory
    hipExtent ext = make_hipExtent(width, height, depth);
    // Make a texture out of the array
    CreateTextureFromVolume(&texRef, &cuArray, bsplineCoeffs, ext, enough_gpu);

    // Dispatch to the appropriate free function, depending on where we allocated it
    if (enough_gpu) {
        hipFree(bsplineCoeffs.ptr);
    } else {
        free(bsplineCoeffs.ptr);
    }
    
    // Get the GPU memory again
    hipMemGetInfo(&free_mem, &total);

    // Figure out how many slices we can do at once
    printf("free_mem: %zu, total: %zu, %u\n", free_mem, total, new_width*new_height*sizeof(float));
    unsigned int slices = ((free_mem >> 1) + (free_mem >> 2))/(sizeof(float)*new_width*new_height);
    slices = (slices > new_depth) ? new_depth : slices;
    printf("slices: %u %u\n", slices, sizeof(float)*new_width*new_height);
    
    // Allocate the output array on the device
    float *output_dev;
    CUDA_SAFE_CALL(hipMalloc(&output_dev, sizeof(float)*new_width*new_height*slices));
    // We need this for the cast functions
    hipPitchedPtr output_pitched = make_hipPitchedPtr((void*)output_dev, new_width*sizeof(float), new_width, new_height);

    // Allocate the host output array
    T *output_host_running = output_array;

    unsigned int depth_offset = 0;

    // Run until we've done everything
    while (depth_offset < new_depth) {

        // Shrink the number of slices if necessary to not go over
        slices = (depth_offset + slices > new_depth) ? (new_depth - depth_offset) : slices;
        printf("depth_offset %d slices %d\n", depth_offset, slices);

        // Calculate the kernel sizes
        dim3 dimBlock(8, 8, 8);
        dim3 dimGrid((new_width+dimBlock.x-1)/dimBlock.x, (new_height+dimBlock.y-1)/dimBlock.y, (slices+dimBlock.z-1)/dimBlock.z);

        // Run the kernel
        transformKernel<<<dimGrid, dimBlock>>>(output_dev, ((float)width)/(float)new_width, ((float)height)/(float)new_height, ((float)depth)/(float)new_depth, new_width, new_height, slices, depth_offset);
        // Make sure everything's done
        hipDeviceSynchronize();
        CUT_CHECK_ERROR("kernel failed");

        depth_offset += slices;
        // Copy from the device and cast to the appropriate data type on the host
        CastVolumeDeviceToHost(output_host_running, output_pitched, new_width, new_height, slices);
        output_host_running += slices*new_width*new_height;
    }

    // Free device arrays
    hipFreeArray(cuArray);
    hipFree(output_dev);
}

extern "C" void interpolate(
        const float *input_array, float *output_array, unsigned int width, unsigned int height, unsigned int depth,
        unsigned int new_width, unsigned int new_height, unsigned int new_depth
    ) {
    interpolate_template(input_array, output_array, width, height, depth, new_width, new_height, new_depth);
}
